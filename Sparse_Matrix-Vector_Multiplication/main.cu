#include "SpMV_CSR.h"
#include "SpMV_COO.h"
#include "../Helper_Code/timer.h"

const float eps = 0.00001;
void checkIfEqual(float* cpuArray, float* gpuArray, unsigned int N){
	for(unsigned int i = 0; i < N; i++) {
        float diff = (cpuArray[i] - gpuArray[i])/cpuArray[i];	//division is to get relative error
        if(diff > eps || diff < -eps) {
            printf("Arrays are not equal (cpuArray[%u] = %e, GPUArray[%u] = %e)\n", i, cpuArray[i], i, gpuArray[i]);
            exit(0);
        }
    }
}

void SpMV_COO_CPU(const COOMatrix<float>& cooMatrix, const float* inVector, float* outVector){
    for(int i = 0; i < cooMatrix.numRows; i++){ outVector[i] = 0; }

    for(int i = 0; i < cooMatrix.numNonzeros; i++){
        unsigned int row = cooMatrix.rowIdxs[i], col = cooMatrix.colIdxs[i];
        outVector[row] += inVector[col] * cooMatrix.values[i];
    }
}

void SpMV_CSR_CPU(const CSRMatrix<float>& csrMatrix, const float* inVector, float* outVector){
    for(int i = 0; i < csrMatrix.numRows; i++){ outVector[i] = 0; }

    for(int row = 0; row < csrMatrix.numRows; row++){
        float sum = 0;
        for(unsigned int i = csrMatrix.rowPtrs[row]; i < csrMatrix.rowPtrs[row + 1]; i++){
            unsigned int col = csrMatrix.colIdxs[i];
            sum += inVector[col] * csrMatrix.values[i];
        }
        outVector[row] = sum;
    }
}

// Multiplies a sparse matrix with a vector
// type 1: uses COO, type 2: uses CSR
int main(int argc, char**argv) {
    hipDeviceSynchronize();

    // Allocate memory and initialize data
    Timer timer;
	unsigned int type = (argc > 1) ? (atoi(argv[1])) : 1;
    unsigned int numNonzeros = (argc > 2) ? (atoi(argv[2])) : 1000000;
    unsigned int numRows = (argc > 3) ? (atoi(argv[3])) : 100000;
    unsigned int numCols = (argc > 4) ? (atoi(argv[4])) : 100000;
	
    if (type == 1){ printf("Running sparse matrix-vector multiplication using the COO format\n"); }
	else { printf("Running sparse matrix-vector multiplication using the CSR format\n"); }
    
    float* inVector = (float*) malloc(numCols*sizeof(float));
    float* outVectorCPU = (float*) malloc(numRows*sizeof(float));
    float* outVectorGPU = (float*) malloc(numRows*sizeof(float));
	
    for(int i = 0; i < numCols; i++){ inVector[i] = 1.0f*rand()/RAND_MAX; }

    COOMatrix<float> cooMatrix(numRows, numCols, numNonzeros, false); 
    CSRMatrix<float> csrMatrix(numRows, numCols, numNonzeros, false);

    if (type == 1) { cooMatrix.generateRandomMatrix(); }
    else { csrMatrix.generateRandomMatrix(); }

    // Compute on CPU
    startTime(&timer);
    if (type == 1){ SpMV_COO_CPU(cooMatrix, inVector, outVectorCPU); }
    else{ SpMV_CSR_CPU(csrMatrix, inVector, outVectorCPU); }
    stopTime(&timer);
    printElapsedTime(timer, "CPU time", BLUE);

	// Compute on GPU
    startTime(&timer);
    if (type == 1){ SpMV_COO_GPU<float>(cooMatrix, inVector, outVectorGPU); }
    else{ SpMV_CSR_GPU<float>(csrMatrix, inVector, outVectorGPU); }
    stopTime(&timer);
    printElapsedTime(timer, "GPU time", RED);

    // Verify result
	checkIfEqual(outVectorCPU, outVectorGPU, numRows);

    // Free memory
    free(inVector);
    free(outVectorCPU); free(outVectorGPU);

    return 0;
}

