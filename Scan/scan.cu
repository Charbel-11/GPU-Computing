#include "hip/hip_runtime.h"
#include "scan.h"
#include "../Helper_Code/timer.h"

#define BLOCK_DIM 1024

// Scans exactly one block
template <typename T>
__global__ void scanKernelKoggeStone(const T* input, T* output, T* partialSums, unsigned int N, bool inclusive){
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ T buffer1_s[BLOCK_DIM], buffer2_s[BLOCK_DIM];
    T* prevBuffer_s = buffer1_s, *curBuffer_s = buffer2_s;
    
    if (inclusive) {
        if (i < N) { prevBuffer_s[threadIdx.x] = input[i]; }
        else {  prevBuffer_s[threadIdx.x] = identity; }
    }
    else {
        if (threadIdx.x > 0 && i - 1 < N) { prevBuffer_s[threadIdx.x] = input[i - 1]; }
        else {  prevBuffer_s[threadIdx.x] = identity; }
    }
    __syncthreads();

    for(unsigned int stride = 1; stride <= BLOCK_DIM/2; stride *= 2){
        curBuffer_s[threadIdx.x] = prevBuffer_s[threadIdx.x] ;
        if (threadIdx.x >= stride){
            curBuffer_s[threadIdx.x] = f<T>(curBuffer_s[threadIdx.x], prevBuffer_s[threadIdx.x - stride]);
        }
        __syncthreads();

        T *tmp = prevBuffer_s;
        prevBuffer_s = curBuffer_s;
        curBuffer_s = tmp;
    }

    if (threadIdx.x == BLOCK_DIM-1){
        if (!inclusive && i < N){ partialSums[blockIdx.x] = f<T>(prevBuffer_s[threadIdx.x], input[i]); }
        else { partialSums[blockIdx.x] = prevBuffer_s[threadIdx.x]; }
    }

    if (i < N) { output[i] = prevBuffer_s[threadIdx.x]; }
}


// Scans exactly one block
template <typename T>
__global__ void scanKernelBrentKung(const T* input, T* output, T* partialSums, unsigned int N, bool inclusive){
    unsigned int segment = 2 * blockIdx.x * blockDim.x;
    unsigned int i1 = segment + threadIdx.x;
    unsigned int i2 = i1 + BLOCK_DIM;

    //We need to store this beforehand for exclusive scans as on recursive scan calls, input = output
    T toAdd = identity;
    if (threadIdx.x == 0 && !inclusive && 2 * BLOCK_DIM - 1 + segment < N){ toAdd = input[segment + 2 * BLOCK_DIM - 1]; }

    __shared__ T buffer_s[2 * BLOCK_DIM];

    if (inclusive){
        if (i1 < N) { buffer_s[threadIdx.x] = input[i1]; }
        else { buffer_s[threadIdx.x] = identity; }
        if (i2 < N) { buffer_s[threadIdx.x + BLOCK_DIM] = input[i2]; }
        else { buffer_s[threadIdx.x + BLOCK_DIM] = identity; }
    }
    else{
        if (threadIdx.x > 0 && i1 - 1 < N) { buffer_s[threadIdx.x] = input[i1 - 1]; }
        else { buffer_s[threadIdx.x] = identity; }
        if (i2 - 1 < N) { buffer_s[threadIdx.x + BLOCK_DIM] = input[i2 - 1]; }
        else { buffer_s[threadIdx.x + BLOCK_DIM] = identity; }
    }
    __syncthreads();

    // Reduction Step
    for (unsigned int stride = 1; stride <= BLOCK_DIM; stride *= 2) {
        unsigned int i = (threadIdx.x + 1) * 2 * stride - 1;  
        if (i < 2 * BLOCK_DIM) { buffer_s[i] = f<T>(buffer_s[i], buffer_s[i - stride]); }
        __syncthreads();
    }

    // Post-reduction Step
    for (unsigned int stride = BLOCK_DIM/2; stride >= 1; stride /= 2) {
        unsigned int i = (threadIdx.x + 1) * 2 * stride - 1;
        if (i + stride < 2 * BLOCK_DIM) { buffer_s[i + stride] = f<T>(buffer_s[i + stride], buffer_s[i]); }
        __syncthreads();
    }

    // Store partial sum
    if (threadIdx.x == 0){
        if (!inclusive && 2 * BLOCK_DIM - 1 + segment < N){ 
            partialSums[blockIdx.x] = f<T>(buffer_s[2 * BLOCK_DIM - 1], toAdd);
        }
        else { partialSums[blockIdx.x] = buffer_s[2 * BLOCK_DIM - 1]; }
    }

    // Store output
    if (i1 < N) { output[i1] = buffer_s[threadIdx.x]; } 
    if (i2 < N) { output[i2] = buffer_s[threadIdx.x + BLOCK_DIM]; }          
}

template <typename T>
__global__ void addKernelKoggeStone(T* output, const T* partialSums, unsigned int N, bool inclusive) {
    unsigned int segment = blockIdx.x * blockDim.x;

    if (blockIdx.x > 0 && segment + threadIdx.x < N) {
        output[segment + threadIdx.x] = f<T>(output[segment + threadIdx.x], partialSums[blockIdx.x - inclusive]);
    }
}

template <typename T>
__global__ void addKernelBrentKung(T* output, const T* partialSums, unsigned int N, bool inclusive) {
    unsigned int segment = 2 * blockIdx.x * blockDim.x;

    if (blockIdx.x > 0) {
        if (segment + threadIdx.x < N) {
            output[segment + threadIdx.x] = f<T>(output[segment + threadIdx.x], partialSums[blockIdx.x - inclusive]);
        }
        if (segment + threadIdx.x + BLOCK_DIM < N) {
            output[segment + threadIdx.x + BLOCK_DIM] = f<T>(output[segment + threadIdx.x + BLOCK_DIM], partialSums[blockIdx.x - inclusive]);
        }
    }
}

template <typename T>
void scanGPUOnDevice(const T* input_d, T* output_d, unsigned int N, unsigned int type, bool inclusive) {
    const unsigned int numThreadsPerBlock = BLOCK_DIM;
    const unsigned int numElementsPerBlock = numThreadsPerBlock * ((type == 1) ? 1 : 2);
    const unsigned int numBlocks = (N + numElementsPerBlock - 1) / numElementsPerBlock;

    // Allocating partial sums
    T *partialSums_d;
    hipMalloc((void**) &partialSums_d, numBlocks*sizeof(T));
    hipDeviceSynchronize();

    // Calling the kernel to scan each block on its own
    if (type == 1) { scanKernelKoggeStone<T> <<< numBlocks, numThreadsPerBlock >>> (input_d, output_d, partialSums_d, N, inclusive); }
    else { scanKernelBrentKung<T> <<< numBlocks, numThreadsPerBlock >>> (input_d, output_d, partialSums_d, N, inclusive); } 

    // Recursively scan partial sums then add
    if (numBlocks > 1) {
        scanGPUOnDevice<T>(partialSums_d, partialSums_d, numBlocks, type, inclusive);
        if (type == 1) { addKernelKoggeStone<T> <<< numBlocks, numThreadsPerBlock >>> (output_d, partialSums_d, N, inclusive); }
        else { addKernelBrentKung<T> <<< numBlocks, numThreadsPerBlock >>> (output_d, partialSums_d, N, inclusive); } 
    }

    // Free memory
    hipFree(partialSums_d);
}

template <typename T>
void scanGPU(const T* input, T* output, unsigned int N, unsigned int type, bool inclusive) {
    Timer timer;

	// Allocating GPU memory
    startTime(&timer);
    T *input_d, *output_d;
    hipMalloc((void**) &input_d, N*sizeof(T));
    hipMalloc((void**) &output_d, N*sizeof(T));
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "GPU Allocation time");

    // Copying data to GPU from Host
    startTime(&timer);
    hipMemcpy(input_d, input, N*sizeof(T), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copying to GPU time");

    // Computing on GPU
    startTime(&timer);
    scanGPUOnDevice<T>(input_d, output_d, N, type, inclusive);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "GPU kernel time", GREEN);

	// Copying data from GPU to Host
    startTime(&timer);
    hipMemcpy(output, output_d, N*sizeof(T), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copying from GPU time");

    // Freeing memory
    startTime(&timer);
    hipFree(input_d); hipFree(output_d);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");
}


template void scanGPU(const double* input, double* output, unsigned int N, unsigned int type, bool inclusive);

template void scanGPUOnDevice(const double* input_d, double* output_d, unsigned int N, unsigned int type, bool inclusive);
template void scanGPUOnDevice(const unsigned int* input_d, unsigned int* output_d, unsigned int N, unsigned int type, bool inclusive);