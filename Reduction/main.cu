#include "common.h"
#include "timer.h"

const float eps = 0.00001;
void checkIfEqual(float cpuVal, float gpuVal){
    float diff = (cpuVal - gpuVal)/cpuVal;	//division is to get relative error
    if(diff > eps || diff < -eps) {
        printf("Values are not equal (cpuVal = %e, gpuVal = %e)\n", cpuVal, gpuVal);
        exit(0);
    }
}

float reduceCPU(float* input, unsigned int N) {
    float sum = identity;
    for(unsigned int i = 0; i < N; ++i) {
        sum = f(sum, input[i]);
    }
    return sum;
}

//Reduces an array A into f(A[1],A[2],...,A[n])
// type 1: usual parallelized reduction;  type 2: uses thread coarsening
int main(int argc, char**argv) {
    hipDeviceSynchronize();

    // Allocate memory and initialize data
    Timer timer;
	unsigned int type = (argc > 1) ? (atoi(argv[1])) : 1;
    unsigned int N = (argc > 2) ? (atoi(argv[2])) : 16000000;
	
	if (type == 1){ printf("Running parallelized reduction\n"); }
	else { printf("Running parallelized reduction with thread coarsening\n"); }
	
    float* input = (float*) malloc(N*sizeof(float));
    for (unsigned int i = 0; i < N; ++i)
        input[i] = 1.0*rand()/RAND_MAX;
    
    // Compute on CPU
    startTime(&timer);
    float cpuVal = reduceCPU(input, N);
    stopTime(&timer);
    printElapsedTime(timer, "CPU time", BLUE);

    // Compute on GPU
    startTime(&timer);
    float gpuVal = reduceGPU(input, N, type);
    stopTime(&timer);
    printElapsedTime(timer, "GPU time", RED);

    // Verify result
	checkIfEqual(cpuVal, gpuVal);

    // Free memory
    free(input);

    return 0;
}

