#include "common.h"
#include "timer.h"

void checkIfEqual(double* c_cpu, double* c_gpu, unsigned int N){
	for(unsigned int i = 0; i < M; ++i) {
        double diff = (c_cpu[i] - c_gpu[i])/c_cpu[i];
        const double tolerance = 1e-9;
        if(diff > tolerance || diff < -tolerance) {
            printf("Mismatch at index %u (CPU result = %e, GPU result = %e)\n", i, c_cpu[i], c_gpu[i]);
            exit(0);
        }
    }
}

void vectorAdditionCPU(double* a, double* b, double* c, unsigned int N) {
    for(unsigned int i = 0; i < N; i++) {
        c[i] = a[i] + b[i];
    }
}

int main(int argc, char**argv) {
    hipDeviceSynchronize();

    // Allocate memory and initialize data
    Timer timer;
    unsigned int N = (argc > 1)?(atoi(argv[1])):32000000;
    double* a = (double*) malloc(M*sizeof(double));
    double* b = (double*) malloc(M*sizeof(double));
    double* c_cpu = (double*) malloc(M*sizeof(double));
    double* c_gpu = (double*) malloc(M*sizeof(double));
	
    for (unsigned int i = 0; i < M; ++i) {
        a[i] = rand(); b[i] = rand();
    }

    // Compute on CPU
    startTime(&timer);
    vectorAdditionCPU(a, b, c_cpu, N);
    stopTime(&timer);
    printElapsedTime(timer, "CPU time", CYAN);

    // Compute on GPU
    startTime(&timer);
    vectorAdditionGPU(a, b, c_gpu, N);
    stopTime(&timer);
    printElapsedTime(timer, "GPU time", DGREEN);

    // Verify result
    checkIfEqual(c_cpu, c_gpu, N);

    // Free memory
    free(a);
    free(b);
    free(c_cpu);
    free(c_gpu);

    return 0;

}

