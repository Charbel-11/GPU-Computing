#include "hip/hip_runtime.h"
#include "SpMV_COO.h"
#include "../Helper_Code/timer.h"

#define BLOCK_DIM 1024

template <typename T>
__global__ void SpMV_COO_Kernel(const COOMatrix<T> cooMatrix, const T* inVector, T* outVector){
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= cooMatrix.numNonzeros){ return; }

    unsigned int row = cooMatrix.rowIdxs[i], col = cooMatrix.colIdxs[i];
    T curVal = cooMatrix.values[i];
    atomicAdd(&outVector[row], inVector[col] * curVal);
}

template <typename T>
void SpMV_COO_GPU(const COOMatrix<T>& cooMatrix, const T* inVector, T* outVector) {
    Timer timer;

	// Allocating GPU memory
    startTime(&timer);
    COOMatrix<T> cooMatrix_d(cooMatrix.numRows, cooMatrix.numCols, cooMatrix.numNonzeros, true);
    cooMatrix_d.allocateArrayMemory();
    T *inVector_d, *outVector_d;
    hipMalloc((void**) &inVector_d, cooMatrix_d.numCols*sizeof(T)); 
    hipMalloc((void**) &outVector_d, cooMatrix_d.numRows*sizeof(T)); 
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "GPU Allocation time");

    //Copying data to GPU from Host
    startTime(&timer);
    hipMemcpy(cooMatrix_d.rowIdxs, cooMatrix.rowIdxs, cooMatrix_d.numNonzeros*sizeof(unsigned int), hipMemcpyHostToDevice); 
    hipMemcpy(cooMatrix_d.colIdxs, cooMatrix.colIdxs, cooMatrix_d.numNonzeros*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(cooMatrix_d.values, cooMatrix.values, cooMatrix_d.numNonzeros*sizeof(T), hipMemcpyHostToDevice);   
    hipMemcpy(inVector_d, inVector, cooMatrix_d.numCols*sizeof(T), hipMemcpyHostToDevice);   
    hipMemset(outVector_d, 0, cooMatrix_d.numRows*sizeof(T));
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copying to GPU time");

    //Calling kernel
    startTime(&timer);
    unsigned int numBlocks = (cooMatrix_d.numNonzeros + BLOCK_DIM - 1) / BLOCK_DIM;
    SpMV_COO_Kernel<T> <<< numBlocks, BLOCK_DIM >>> (cooMatrix_d, inVector_d, outVector_d);    
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "GPU kernel time", GREEN);
	
	//Copying data from GPU to Host
    startTime(&timer);
    hipMemcpy(outVector, outVector_d, cooMatrix_d.numRows*sizeof(T), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copying from GPU time");

	//Freeing GPU memory
    startTime(&timer);
    hipFree(inVector_d); hipFree(outVector_d);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "GPU Deallocation time");
}

template void SpMV_COO_GPU(const COOMatrix<float>& cooMatrix, const float* inVector, float* outVector);