#include "hip/hip_runtime.h"
#include "common.h"
#include "timer.h"

#define BLOCK_DIM 32

#define cudaErrorCheck(error) { gpuAssert((error), __FILE__, __LINE__); }
void gpuAssert(hipError_t code, const char *file, const int line) {
    if (code != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s in file %s at line %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

// A (MxK) * B (KxN) = C (MxN)
__global__ void matrixMultiplicationKernel(float* A, float* B, float* C, unsigned int M, unsigned int N, unsigned int K) {
    unsigned int outRow = (blockIdx.y * blockDim.y) + threadIdx.y;
    unsigned int outColumn = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (outRow < M && outColumn < N) {
        float sum = 0.0f;		// Register tiling
        for(unsigned int i = 0; i < K; i++) {
            sum += A[outRow*K + i] * B[i*N + outColumn];
        }
        C[outRow*N + outColumn] = sum;
    }
}

__global__ void matrixMultiplicationKernelWithTiling(float* A, float* B, float* C, unsigned int M, unsigned int N, unsigned int K) {
    unsigned int outRow = (blockIdx.y * blockDim.y) + threadIdx.y;
    unsigned int outColumn = (blockIdx.x * blockDim.x) + threadIdx.x;
	float sum = 0.0f;		
	
	__shared__ float A_s[BLOCK_DIM][BLOCK_DIM];
	__shared__ float B_s[BLOCK_DIM][BLOCK_DIM];	
	
	for(unsigned int tileIdx = 0; tileIdx < (K + BLOCK_DIM - 1) / BLOCK_DIM; tileIdx++){
		if (outRow < M && (tileIdx*BLOCK_DIM + threadIdx.x) < K) {
			A_s[threadIdx.y][threadIdx.x] = A[outRow*K + (tileIdx*BLOCK_DIM + threadIdx.x)];
		}
		else { A_s[threadIdx.y][threadIdx.x] = 0.0; }
		
		if ((tileIdx*BLOCK_DIM + threadIdx.y) < K && outColumn < N) {
			B_s[threadIdx.y][threadIdx.x] = B[(tileIdx*BLOCK_DIM + threadIdx.y)*N + outColumn];
		}
		else { B_s[threadIdx.y][threadIdx.x] = 0.0; }
		
		__syncthreads();
		
		if (outRow < M && outColumn < N) {
			for(unsigned int i = 0; i < BLOCK_DIM; i++) {
				sum += A_s[threadIdx.y][i] * B_s[i][threadIdx.x];
			}
		}
		__syncthreads();
	} 
	
	if (outRow < M && outColumn < N) { C[outRow*N + outColumn] = sum; }
}

void matrixMultiplicationGPU(float* A, float* B, float* C, unsigned int M, unsigned int N, unsigned int K, unsigned int type) {
    Timer timer;

    //Allocating GPU memory
    startTime(&timer);

    float *A_d, *B_d, *C_d;
    hipError_t errMallocA = hipMalloc((void **) &A_d, M * K * sizeof(float)); cudaErrorCheck(errMallocA);
    hipError_t errMallocB = hipMalloc((void **) &B_d, K * N * sizeof(float)); cudaErrorCheck(errMallocB);
    hipError_t errMallocC = hipMalloc((void **) &C_d, M * N * sizeof(float)); cudaErrorCheck(errMallocC);

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "GPU Allocation time");

    //Copying data to GPU from Host
    startTime(&timer);

    hipError_t errMemcpyA = hipMemcpy(A_d, A, M * K * sizeof(float), hipMemcpyHostToDevice); cudaErrorCheck(errMemcpyA);
    hipError_t errMemcpyB = hipMemcpy(B_d, B, K * N * sizeof(float), hipMemcpyHostToDevice); cudaErrorCheck(errMemcpyB);

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copying to GPU time");

    //Calling kernel
    startTime(&timer);

    dim3 numThreadsPerBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 numBlocks((N + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x, (M + numThreadsPerBlock.y - 1) / numThreadsPerBlock.y);
    if (type == 1) { matrixMultiplicationKernel<<< numBlocks, numThreadsPerBlock >>>(A_d, B_d, C_d, M, N, K); }
	else { matrixMultiplicationKernelWithTiling<<< numBlocks, numThreadsPerBlock >>>(A_d, B_d, C_d, M, N, K); }
    cudaErrorCheck(hipGetLastError());			//For arguments errors
    cudaErrorCheck(hipDeviceSynchronize());	//For execution error in the kernel

    stopTime(&timer);
    printElapsedTime(timer, "Running the kernel time", GREEN);

    //Copying data from GPU to Host
    startTime(&timer);

    hipError_t errMemcpyC = hipMemcpy(C, C_d, M * N * sizeof(float), hipMemcpyDeviceToHost); cudaErrorCheck(errMemcpyC);

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copying from GPU time");

    //Freeing GPU memory
    startTime(&timer);

    hipError_t errFreeA = hipFree(A_d); cudaErrorCheck(errFreeA);
    hipError_t errFreeB = hipFree(B_d); cudaErrorCheck(errFreeB);
    hipError_t errFreeC = hipFree(C_d); cudaErrorCheck(errFreeC);

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "GPU Deallocation time");
}

