#include "hip/hip_runtime.h"
#include "common.h"
#include "timer.h"

/*
 * Macro to avoid duplication of error checking code for functions
 * such as hipMalloc(), hipMemcpy() and hipFree()
 */
#define cudaErrorCheck(error) { gpuAssert((error), __FILE__, __LINE__); }

/*
 * Abort is set to True by default in order to immediately stop program execution
 */ 
void gpuAssert(hipError_t code, const char *file, const int line, bool abort=true) {
   
    if (code != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s in file: %s (line: %d)\n", hipGetErrorString(code), file, line);
      if (abort) {
          exit(code);
      }
   }

}

__global__ void vectorAdditionKernel(double* a, double* b, double* c, unsigned int N) {
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;

	if (i < N){
		c[i] = a[i] + b[i];
    }

}

void vectorAdditionGPU(double* a, double* b, double* c, unsigned int M) {

    Timer timer;

    /*
     * Allocate GPU memory
     */
    startTime(&timer);

    double *a_d, *b_d, *c_d;
    hipError_t errMallocA = hipMalloc((void **) &a_d, M * sizeof(double));
    cudaErrorCheck(errMallocA);

    hipError_t errMallocB = hipMalloc((void **) &b_d, M * sizeof(double));
    cudaErrorCheck(errMallocB);

    hipError_t errMallocC = hipMalloc((void **) &c_d, M * sizeof(double));
    cudaErrorCheck(errMallocC);

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    /*
     * Copy data to GPU from Host
     */
    startTime(&timer);

    hipError_t errMemcpyA = hipMemcpy(a_d, a, M * sizeof(double), hipMemcpyHostToDevice);
    cudaErrorCheck(errMemcpyA);

    hipError_t errMemcpyB = hipMemcpy(b_d, b, M * sizeof(double), hipMemcpyHostToDevice);
    cudaErrorCheck(errMemcpyB);

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");
    
    /* 
     * Call kernel
     */
    startTime(&timer);

    const unsigned int numThreadsPerBlock = 512;
    const unsigned int numBlocks = (M + numThreadsPerBlock - 1) / numThreadsPerBlock;
    vecMax_kernel<<< numBlocks, numThreadsPerBlock >>>(a_d, b_d, c_d, M);

    /*
     * Call hipGetLastError() first in order to check for any argument errors in the kernel
     *  
     * Call error-checking macro on hipDeviceSynchronize() afterwards in order to wait for the
     * kernel to completely finish and check for any error while executing the kernel code
     */ 
    cudaErrorCheck(hipGetLastError());
    cudaErrorCheck(hipDeviceSynchronize());
    
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);
    
    /*
     * Copy data from GPU to Host
     */
    startTime(&timer);

    hipError_t errMemcpyC = hipMemcpy(c, c_d, M * sizeof(double), hipMemcpyDeviceToHost);
    cudaErrorCheck(errMemcpyC);

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");
    
    /*
     * Free GPU memory
     */
    startTime(&timer);

    hipError_t errFreeA = hipFree(a_d);
    cudaErrorCheck(errFreeA);

    hipError_t errFreeB = hipFree(b_d);
    cudaErrorCheck(errFreeB);

    hipError_t errFreeC = hipFree(c_d);
    cudaErrorCheck(errFreeC);

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

}