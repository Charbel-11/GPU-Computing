#include "hip/hip_runtime.h"
#include "common.h"
#include "timer.h"

#define cudaErrorCheck(error) { gpuAssert((error), __FILE__, __LINE__); }
void gpuAssert(hipError_t code, const char *file, const int line) {
    if (code != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s in file %s at line %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

__global__ void vectorAdditionKernel(double* a, double* b, double* c, unsigned int N) {
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (i < N){ c[i] = a[i] + b[i]; }
}

__global__ void vectorMaxKernel(double* a, double* b, double* c, unsigned int N) {
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (i < N){ c[i] = (a[i] > b[i]) ? a[i] : b[i]; }
}

__global__ void vectorProductKernel(double* a, double* b, double* c, unsigned int N) {
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (i < N){ c[i] = a[i] * b[i]; }
}

void vectorOperationGPU(double* a, double* b, double* c, unsigned int N, unsigned int type) {
    Timer timer;

    //Allocating GPU memory
    startTime(&timer);

    double *a_d, *b_d, *c_d;
    hipError_t errMallocA = hipMalloc((void **) &a_d, N * sizeof(double)); cudaErrorCheck(errMallocA);
    hipError_t errMallocB = hipMalloc((void **) &b_d, N * sizeof(double)); cudaErrorCheck(errMallocB);
    hipError_t errMallocC = hipMalloc((void **) &c_d, N * sizeof(double)); cudaErrorCheck(errMallocC);

    hipDeviceSynchronize();		//To get the correct time since GPU/CPU run asynchronously
    stopTime(&timer);
    printElapsedTime(timer, "GPU Allocation time");

    //Copying data to GPU from Host
    startTime(&timer);

    hipError_t errMemcpyA = hipMemcpy(a_d, a, N * sizeof(double), hipMemcpyHostToDevice); cudaErrorCheck(errMemcpyA);
    hipError_t errMemcpyB = hipMemcpy(b_d, b, N * sizeof(double), hipMemcpyHostToDevice); cudaErrorCheck(errMemcpyB);

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copying to GPU time");
    
    //Calling kernel
    startTime(&timer);

    const unsigned int numThreadsPerBlock = 512;
    const unsigned int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;
    if (type == 1) { vectorAdditionKernel<<< numBlocks, numThreadsPerBlock >>>(a_d, b_d, c_d, N); }
    else if (type == 2) { vectorMaxKernel<<< numBlocks, numThreadsPerBlock >>>(a_d, b_d, c_d, N); }
    else if (type == 3) { vectorProductKernel<<< numBlocks, numThreadsPerBlock >>>(a_d, b_d, c_d, N); }
    cudaErrorCheck(hipGetLastError());			//For arguments errors
    cudaErrorCheck(hipDeviceSynchronize());	//For execution error in the kernel
    
    stopTime(&timer);
    printElapsedTime(timer, "Running the kernel time", GREEN);
    
    //Copying data from GPU to Host
    startTime(&timer);

    hipError_t errMemcpyC = hipMemcpy(c, c_d, N * sizeof(double), hipMemcpyDeviceToHost);  cudaErrorCheck(errMemcpyC);

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copying from GPU time");
    
    //Freeing GPU memory
    startTime(&timer);

    hipError_t errFreeA = hipFree(a_d); cudaErrorCheck(errFreeA);
    hipError_t errFreeB = hipFree(b_d); cudaErrorCheck(errFreeB);
    hipError_t errFreeC = hipFree(c_d); cudaErrorCheck(errFreeC);
    
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "GPU Deallocation time");
}
